
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>



void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg,  hipGetErrorString( err) );
    }                         
}

hipError_t FFTWithCuda(float *input, float *output, size_t size);

__global__ void addKernel(float *input, float *output)
{
	int i = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	output[i] = input[i] + 0.5;
	if(output[i] > 1)
		output[i] = 1;
}

int applyFFT(float* input, float* output, int size)
{

	// Add vectors in parallel.
	hipError_t cudaStatus = FFTWithCuda(input, output, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "FFTWithCuda failed!");
		return 1;
	}

	// cudaDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FFTWithCuda(float *input, float *output, size_t size)
{
	int dimElement = size;
	int numThreadsPerBlock = 100; 
	int numBlocks = dimElement / numThreadsPerBlock;
	dim3 dimGrid(numBlocks);
	dim3 dimBlock(numThreadsPerBlock);
	size_t memSize = dimElement * sizeof(*input);
	printf("nb of elements: %d \n", dimElement);
	printf("memory size: %d \n", memSize);

	float *dev_i = 0;
	float *dev_o = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		checkCUDAError("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_i, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		checkCUDAError("hipMalloc failed!");
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_o, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		checkCUDAError("hipMalloc failed!");
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_i, input, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		checkCUDAError("hipMemcpy failed!");
		return cudaStatus;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel<<<dimGrid, dimBlock>>>(dev_i, dev_o);

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		checkCUDAError("hipDeviceSynchronize failed!");
		printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(output, dev_o, size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		checkCUDAError("hipMemcpy failed!");
		return cudaStatus;
	}

	return cudaStatus;
}
